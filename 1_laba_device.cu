#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>
#include <string>

#define VECTOR_SIZE 10000000
#define CHECK_CUDA_ERROR(a) {                                                   \
    hipError_t err = a;                                                        \
    if (err != hipSuccess) {                                                   \
        std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__    \
                  << ": " << hipGetErrorString(err) << std::endl;              \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}

__global__ void sumvec_d(double* a, double* b, double *c, int N) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id < N) {
        c[global_id] = sin(a[global_id])*sin(a[global_id]) + cos(b[global_id])*cos(b[global_id]);
    }
}


int main(int argc, char** argv) {
    size_t N;
    if (argc > 1)
        N = atoi(argv[1]);
    else 
        N = VECTOR_SIZE;
        
    double *h_a, *h_b, *h_c;
    double *d_a, *d_b, *d_c;

    h_a = (double*)malloc(N*sizeof(double));
    h_b = (double*)malloc(N*sizeof(double));
    h_c = (double*)malloc(N*sizeof(double));

    memset(h_c, 0, N);
    for (int i = 0; i < N; ++i) {
        h_a[i] = 1.0;
        h_b[i] = 1.0;
    }

    CHECK_CUDA_ERROR(hipMalloc((void**)&d_a, N * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_b, N * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_c, N * sizeof(double)));
    CHECK_CUDA_ERROR(hipMemset(d_a, 0, N * sizeof(double)));

    hipEvent_t t1, t2;
    CHECK_CUDA_ERROR(hipEventCreate(&t1));
    CHECK_CUDA_ERROR(hipEventCreate(&t2));

    hipEvent_t copy_start_1, copy_stop_1;
    hipEvent_t copy_start_2, copy_stop_2;
    CHECK_CUDA_ERROR(hipEventCreate(&copy_start_1));
    CHECK_CUDA_ERROR(hipEventCreate(&copy_stop_1));
    CHECK_CUDA_ERROR(hipEventCreate(&copy_start_2));
    CHECK_CUDA_ERROR(hipEventCreate(&copy_stop_2));

    CHECK_CUDA_ERROR(hipEventRecord(copy_start_1));
    CHECK_CUDA_ERROR(hipMemcpy(d_a, h_a, N * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_b, h_b, N * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipEventRecord(copy_stop_1));

    int threads = 128;
    int blocks = N / threads + 1;
    CHECK_CUDA_ERROR(hipEventRecord(t1));
    sumvec_d<<<blocks, threads>>>(d_a, d_b, d_c, N);
    CHECK_CUDA_ERROR(hipEventRecord(t2));
    CHECK_CUDA_ERROR(hipEventRecord(copy_start_1));
    CHECK_CUDA_ERROR(hipMemcpy(h_c, d_c, N * sizeof(double), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipEventRecord(copy_stop_2));
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    float t_kernel = 0, t_copy_1 = 0, t_copy_2 = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&t_kernel, t1, t2));  
    CHECK_CUDA_ERROR(hipEventElapsedTime(&t_copy1, copy_start_1, copy_stop_1)); 
    CHECK_CUDA_ERROR(hipEventElapsedTime(&t_copy2, copy_start_2, copy_stop_1)); 
    std::cout << "t_kernel: " << t_kernel  << " ms" << std::endl;
    std::cout << "t_copy: " << t_copy_1 + t_copy_2 << " ms" << std::endl;

    free(h_a);
    free(h_b);
    free(h_c);

    CHECK_CUDA_ERROR(hipFree(d_a));
    CHECK_CUDA_ERROR(hipFree(d_b));
    CHECK_CUDA_ERROR(hipFree(d_c));

    CHECK_CUDA_ERROR(hipEventDestroy(t1));
    CHECK_CUDA_ERROR(hipEventDestroy(t2));
    CHECK_CUDA_ERROR(hipEventDestroy(copy_start));
    CHECK_CUDA_ERROR(hipEventDestroy(copy_stop));

    return 0;
}
