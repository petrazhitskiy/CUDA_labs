#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <chrono>

#define VECTOR_SIZE 10000000
#define STREAM_NUM 2

#define CHECK_CUDA_ERROR(call) {                                        \
    hipError_t err = call;                                             \
    if (err != hipSuccess) {                                           \
        std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__ \
                  << ": " << hipGetErrorString(err) << std::endl;      \
        exit(EXIT_FAILURE);                                             \
    }                                                                   \
}

__global__ void sumvec_d(double* a, double* b, double *c, int N) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_id < N) {
        c[global_id] = sin(a[global_id])*sin(a[global_id]) + cos(b[global_id])*cos(b[global_id]);
    }
}


int main(int argc, char** argv) {
    size_t N;
    size_t numStreams; 
    if (argc > 2){
        N = atoi(argv[1]);
        numStreams = atoi(argv[2]);
    }
    else if (argc > 1){	    
        N = atoi(argv[1]);
	    numStreams = STREAM_NUM;
    }
    else {
        N = VECTOR_SIZE;
        numStreams= STREAM_NUM;
    }

    double *h_a, *h_b, *h_c;
    double *d_a, *d_b, *d_c;

    h_a = (double*)malloc(N*sizeof(double));
    h_b = (double*)malloc(N*sizeof(double));
    h_c = (double*)malloc(N*sizeof(double));

    memset(h_c, 0, N);
    for (int i = 0; i < N; ++i) {
        h_a[i] = 1.0;
        h_b[i] = 1.0;
    }
    
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_a, N * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_b, N * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_c, N * sizeof(double)));
    CHECK_CUDA_ERROR(hipMemset(d_a, 0, N * sizeof(double)));

    hipStream_t streams[numStreams];
    for (int i = 0; i < numStreams; ++i) {
        CHECK_CUDA_ERROR(hipStreamCreate(&streams[i]));
    }

    int segmentSize = N / numStreams;

    hipEvent_t t1, t2;
    CHECK_CUDA_ERROR(hipEventCreate(&t1));
    CHECK_CUDA_ERROR(hipEventCreate(&t2));
    hipEvent_t copy_start, copy_stop;
    CHECK_CUDA_ERROR(hipEventCreate(&copy_start));
    CHECK_CUDA_ERROR(hipEventCreate(&copy_stop));
    CHECK_CUDA_ERROR(hipEventRecord(copy_start));
    for (int i = 0; i < numStreams; ++i) {
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_a + i * segmentSize, h_a + i * segmentSize, segmentSize * sizeof(double), hipMemcpyHostToDevice, streams[i]));
        CHECK_CUDA_ERROR(hipMemcpyAsync(d_b + i * segmentSize, h_b + i * segmentSize, segmentSize * sizeof(double), hipMemcpyHostToDevice, streams[i]));
    }
    CHECK_CUDA_ERROR(hipEventRecord(copy_stop));
    int threads = 128;
    int blocks = (segmentSize + threads - 1) / threads;
    CHECK_CUDA_ERROR(hipEventRecord(t1));
    for (int i = 0; i < numStreams; ++i) {
        sumvec_d<<<blocks, threads, 0, streams[i]>>>(d_a + i * segmentSize, d_b + i * segmentSize, d_c + i * segmentSize, segmentSize);
    }
    CHECK_CUDA_ERROR(hipEventRecord(t2));
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    for (int i = 0; i < numStreams; ++i) {
        CHECK_CUDA_ERROR(hipMemcpyAsync(h_c + i * segmentSize, d_c + i * segmentSize, segmentSize * sizeof(double), hipMemcpyDeviceToHost, streams[i]));
    }
    float t_kernel = 0, t_copy = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&t_kernel, t1, t2));  
    CHECK_CUDA_ERROR(hipEventElapsedTime(&t_copy, copy_start, copy_stop)); 

    std::cout << "t_kernel: " << t_kernel << " ms" << std::endl;
    std::cout << "t_copy: " << t_copy << " ms" << std::endl;

    for (int i = 0; i < numStreams; ++i) {
        CHECK_CUDA_ERROR(hipStreamSynchronize(streams[i]));
    }
    CHECK_CUDA_ERROR(hipFree(d_a));
    CHECK_CUDA_ERROR(hipFree(d_b));
    CHECK_CUDA_ERROR(hipFree(d_c));

    for (int i = 0; i < numStreams; ++i) {
        CHECK_CUDA_ERROR(hipStreamDestroy(streams[i]));
    }

    return 0;
}
